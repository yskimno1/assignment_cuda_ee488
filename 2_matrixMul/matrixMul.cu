#include <stdio.h>
#include <assert.h>
#include <string.h>
#include <sstream>
#include <fstream>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <unistd.h>
#include <time.h>
#include <pthread.h>
#include <cmath>

#include <hip/hip_runtime.h>

//#define _DEBUG

// helpful macros
#define FatalError(s) {                                                \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;\
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(EXIT_FAILURE);                                                \
}


#define CheckCudaErrors(status) {                                      \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      assert(0);                                                        \
      FatalError(_error.str());                                        \
    }                                                                  \
}

#define BLOCK_SIZE	32


void computeRefMatrixMul(float *C, const float *A, const float *B, unsigned int height_A, unsigned int width_A, unsigned int width_B) {
	for(unsigned int i=0; i<height_A; i++) {
		for(unsigned int j=0; j<width_B; j++) {
	    	double sum = 0;
            for(unsigned int k=0; k<width_A; k++) {
				double a = A[(i*width_A)+k];
				double b = B[(k*width_B)+j];
				sum += a*b;
            }
            C[(i*width_B)+j] = (float)sum;
        }
	}
}

__global__ 
void matrixMul_naive(float* C, float* A, float* B, int wA, int wB) {
    // TODO: fill me
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float Pvalue = 0;
    if((row < wA) && (col < wB)){
        for(int k=0; k<wA; k++){
            Pvalue = Pvalue + A[(row*wA)+k] * B[(k*wB)+col];
        }
        C[row*wB+col] = Pvalue; 
    }
}

__global__ 
void matrixMul_shmem( float* C, float* A, float* B, int wA, int wB)
{
    // TODO: fill me
    __shared__ float ds_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float ds_B[BLOCK_SIZE][BLOCK_SIZE]; 

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    float Pvalue = 0;

    for(int p=0; p<wA/BLOCK_SIZE; p++){
        ds_A[ty][tx] = A[row*wA + p*BLOCK_SIZE+tx];
        ds_B[ty][tx] = B[(p*BLOCK_SIZE+ty)*wB+col];
        __syncthreads();

        for(int i=0; i<BLOCK_SIZE; i++){
            Pvalue += ds_A[ty][i] * ds_B[i][tx];
        }
        __syncthreads();
    }
    C[row*wB+col] = Pvalue; 
}

void randomInitialization(float *data, int size) {
	srand(time(NULL));
	for(int i=0; i<size; i++) {
		data[i] = rand()/(float)RAND_MAX;
  }
}

bool compareArray(const float *reference, const float *data, const unsigned int len, const float epsilon) {
  assert(epsilon >= 0);
  float error = 0;
  float ref = 0;
  
  for(unsigned int i=0; i<len; i++) {
    float diff = reference[i] - data[i];
    error += diff * diff;
    ref += reference[i] * reference[i];
  }
  float normRef = sqrtf(ref);

  if (fabs(ref) < 1e-7) {
#ifdef _DEBUG
		std::cerr << "ERROR, reference l2-norm is 0\n";
#endif
      return false;
  }
  float normError = sqrtf(error);
  error = normError / normRef;
  bool result = error < epsilon;
#ifdef _DEBUG
  if (! result) {
      std::cerr<<"ERROR, l2-norm error "<<error<<" is greater than epsilon "<<epsilon<<"\n";
  }
#endif
  return result;
}

int matrixMul(int block_size, dim3 &dimA, dim3 &dimB)
{
  // Allocate host memory for matrices A
  unsigned int size_A		= dimA.x*dimA.y;
  unsigned int mem_size_A = sizeof(float)*size_A;
  float *h_A = (float*)malloc(mem_size_A);

  // Allocate host memory for matrices B
  unsigned int size_B		= dimB.x*dimB.y;
  unsigned int mem_size_B = sizeof(float)*size_B;
  float *h_B = (float*)malloc(mem_size_B);

  // Allocate host matrix C
  dim3 dimC(dimB.x, dimA.y, 1);
  unsigned int mem_size_C = dimC.x*dimC.y*sizeof(float);
  unsigned int size_C		= dimC.x*dimC.y;
  float *h_C = (float*)malloc(mem_size_C);

  // Initialize host memory A & B 
  randomInitialization(h_A, size_A);
  randomInitialization(h_B, size_B);
  randomInitialization(h_C, size_C);

  // compute gold solution
  printf("\n[Step-1] Computing reference result using host-side CPU ... ");
  float *reference_C = (float *)malloc(mem_size_C);
  computeRefMatrixMul(reference_C, h_A, h_B, dimA.y, dimA.x, dimB.x);
  printf("DONE!\n");

  // Allocate device memory for A, B, and C
  float *d_A, *d_B, *d_C;
  CheckCudaErrors(hipMalloc((void **)&d_A, mem_size_A));
  CheckCudaErrors(hipMalloc((void **)&d_B, mem_size_B));
  CheckCudaErrors(hipMalloc((void **)&d_C, mem_size_C));

  // copy host-side A and B to device
  CheckCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
  CheckCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));
  CheckCudaErrors(hipMemcpy(d_C, h_C, mem_size_B, hipMemcpyHostToDevice));

  // Setup execution parameters
  dim3 threads(block_size, block_size);
  dim3 grid((dimB.x/threads.x), (dimA.y/threads.y));
    
	//----------------------------------------------------------
	// Part A. Naive implementation of matrix-multiplication
	//----------------------------------------------------------
  // Allocate CUDA events that is used for measuring kernel execution latency
  hipEvent_t start, stop;
  CheckCudaErrors(hipEventCreate(&start));
  CheckCudaErrors(hipEventCreate(&stop));
    
	// For accurate performance measurements, perform a dummy kernel launch for warm-up
  matrixMul_naive<<< grid, threads >>>(d_C, d_A, d_B, dimA.x, dimB.x);
  hipDeviceSynchronize();
  
  printf("\n[Step-2] Computing result using naive version of CUDA kernel ... ");

  // Record the start event
  CheckCudaErrors(hipEventRecord(start, NULL));

  // Execute the kernel
  int nIter = 500;
  for(int j=0; j<nIter; j++) {
		matrixMul_naive<<<grid, threads>>>(d_C, d_A, d_B, dimA.x, dimB.x);
  }

  // Record the stop event
  CheckCudaErrors(hipEventRecord(stop, NULL));
  // Wait for the stop event to be finalized
  CheckCudaErrors(hipEventSynchronize(stop));
  printf("DONE!\n");

	// measure average latency incurred for this kernel execution
  float msecTotal = 0.0f;
  CheckCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  // Compute and print the performance
  float msecPerMatrixMul		= msecTotal/nIter;
  double flopsPerMatrixMul	= 2.0*(double)dimA.x*(double)dimA.y*(double)dimB.x;
  double gigaFlops			= (flopsPerMatrixMul*1.0e-9f)/(msecPerMatrixMul/1000.0f);
  printf("- Math Size = %.0f OPs\n", flopsPerMatrixMul);
  printf("- Performance = %.2f GFLOP/sec (Time = %.3f msec)\n", gigaFlops, msecPerMatrixMul); 

  // Copy result from device to host
  CheckCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

  printf("- Correctness (reference vs. CUDA): ");
	bool correct = compareArray(reference_C, h_C, size_C, 1.0e-6f); 
  // check result
	if(correct != true) {
		printf("%s\n", "FAIL");
	}
	else {
		printf("%s\n", "PASS");
	}

	//----------------------------------------------------------
	// Part B. Better implementation of matrix-multiplication
	//----------------------------------------------------------
  // Initialize host memory A & B 
  randomInitialization(h_A, size_A);
  randomInitialization(h_B, size_B);
  randomInitialization(h_C, size_C);
  // compute gold solution
  computeRefMatrixMul(reference_C, h_A, h_B, dimA.y, dimA.x, dimB.x);

  // copy host-side A and B to device
  CheckCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
	CheckCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));
	CheckCudaErrors(hipMemcpy(d_C, h_C, mem_size_B, hipMemcpyHostToDevice));

	// For accurate performance measurements, perform a dummy kernel launch for warm-up
  matrixMul_shmem<<< grid, threads >>>(d_C, d_A, d_B, dimA.x, dimB.x);
  hipDeviceSynchronize();
  
  printf("\n[Step-3] Computing result using shmem version of CUDA kernel ... ");

  // Record the start event
  CheckCudaErrors(hipEventRecord(start, NULL));

  // Execute the kernel
  nIter = 500;
  for(int j=0; j<nIter; j++) {
		matrixMul_shmem<<<grid, threads>>>(d_C, d_A, d_B, dimA.x, dimB.x);
  }

  // Record the stop event
  CheckCudaErrors(hipEventRecord(stop, NULL));
  // Wait for the stop event to be finalized
  CheckCudaErrors(hipEventSynchronize(stop));
  printf("DONE!\n");

	// measure average latency incurred for this kernel execution
  msecTotal = 0.0f;
  CheckCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  // Compute and print the performance
  msecPerMatrixMul	= msecTotal/nIter;
  flopsPerMatrixMul	= 2.0*(double)dimA.x*(double)dimA.y*(double)dimB.x;
  gigaFlops			= (flopsPerMatrixMul*1.0e-9f)/(msecPerMatrixMul/1000.0f);
  printf("- Math Size = %.0f OPs\n", flopsPerMatrixMul);
  printf("- Performance = %.2f GFLOP/sec (Time = %.3f msec)\n", gigaFlops, msecPerMatrixMul); 

  // Copy result from device to host
  CheckCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

  printf("- Correctness (reference vs. CUDA): ");
	correct = compareArray(reference_C, h_C, size_C, 1.0e-6f); 
  // check result
	if(correct != true) {
		printf("%s\n", "FAIL");
	}
	else {
		printf("%s\n", "PASS");
	}

  // Clean up memory
  free(h_A);
  free(h_B);
  free(h_C);
  CheckCudaErrors(hipFree(d_A));
  CheckCudaErrors(hipFree(d_B));
	CheckCudaErrors(hipFree(d_C));
	//CheckCudaErrors(cudaFree(d_C));	// This should cause an error ...

  if(correct) {
      return EXIT_SUCCESS;
  }
  else
  {
      return EXIT_FAILURE;
  }
}

// main
int main(int argc, char **argv)
{
	printf("\n---------------------------------------------------\n");
  printf("[Lab 1] Part 2: Matrix-Multiplication Using CUDA\n");
	printf("---------------------------------------------------\n");

	// dimension of matrix A and B
  dim3 dimA(10*BLOCK_SIZE, 10*BLOCK_SIZE, 1);
  dim3 dimB(20*BLOCK_SIZE, 10*BLOCK_SIZE, 1);

	// check if dimension of A & B match properly
  if(dimA.x!=dimB.y) {
	printf("Error: outer matrix dimensions must be equal. (%d != %d)\n", dimA.x, dimB.y);
	exit(EXIT_FAILURE);
  }
	// target matrix configuration
  printf("\n- MatrixA(%d,%d), MatrixB(%d,%d)\n", dimA.x, dimA.y, dimB.x, dimB.y);

	// do matrix multiplication
  exit(matrixMul(BLOCK_SIZE, dimA, dimB));
}
