#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);

#define DEBUG
#ifdef DEBUG
 
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
        hipGetErrorString(code), file, line);
        if (abort) exit(code);    
    }
}
#else
#define cudaCheckError(ans) ans

#endif

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

void
saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {
    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    int size = N *sizeof(float);
    //
    // TODO: allocate device memory buffers on the GPU using
    // hipMalloc.  The started code issues warnings on build because
    // these buffers are used in the call to saxpy_kernel below
    // without being initialized.
    //

    hipMalloc(&device_x, size);
    hipMalloc(&device_y, size);
    hipMalloc(&device_result, size);

    // start timing after allocation of device memory.
    double startTime = CycleTimer::currentSeconds();

    //
    // TODO: copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_x, xarray, size, hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, size, hipMemcpyHostToDevice);

    //
    // TODO: insert time here to begin timing only the kernel
    //
    double kernel_startTime = CycleTimer::currentSeconds();

    // run saxpy_kernel on the GPU
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);

    //
    // TODO: insert timer here to time only the kernel.  Since the
    // kernel will run asynchronously with the calling CPU thread, you
    // need to call hipDeviceSynchronize() before your timer to
    // ensure the kernel running on the GPU has completed.  (Otherwise
    // you will incorrectly observe that almost no time elapses!)
    //
    hipDeviceSynchronize();
    double kernel_endTime = CycleTimer::currentSeconds();

    //
    // TODO: copy result from GPU using hipMemcpy
    //
    hipMemcpy(resultarray, device_result, size, hipMemcpyDeviceToHost);
    
    // end timing after result has been copied back into host memory.
    // The time elapsed between startTime and endTime is the total
    // time to copy data to the GPU, run the kernel, and copy the
    // result back to the CPU
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double kernelDuration = kernel_endTime - kernel_startTime;
    printf("Overall time: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    printf("Kernel time: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * kernelDuration, toBW(totalBytes, kernelDuration));
    //
    // TODO free memory buffers on the GPU
    //
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);

    // below code calculates the time that sequential CPU-based SAXPY used
    // float* resultarray_CPU = new float[N];
    // double startTime_CPU = CycleTimer::currentSeconds();
    // for (int i=0; i<N; i++) {
    //     resultarray_CPU[i] = xarray[i]*alpha+yarray[i];
    // }
    // double endTime_CPU = CycleTimer::currentSeconds();
    // double CPUDuration = endTime_CPU - startTime_CPU;
    // printf("CPU time: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * CPUDuration, toBW(totalBytes, CPUDuration));
    // delete [] resultarray_CPU;
}
void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
